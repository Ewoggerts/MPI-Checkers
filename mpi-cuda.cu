#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <iostream>
#include <mpi_checker_recursive_captures>

#define THREADS_PER_BLOCK 64  // Define the number of threads per block

extern "C" 
{
    void runCudaAnalysis(BoardList *boards,  int *winners);
}

// CUDA kernel to count points inside the unit circle
__global__ void analyze(BoardList *boards, int *winner) {
    int local_tid = threadIdx.x;
    int boardIdx = blockIdx.x;

    int row = local_tid / 8;
    int col = local_tid % 8;

    __shared__ int sharedRed;
    __shared__ int sharedBlack;

    if (local_tid == 0) {
        sharedRed = 0;
        sharedBlack = 0;
    }
    __syncthreads();

    Board current_board = boards->board[boardIdx];
    char piece = current_board.board[row][col];

    if (piece == 'r' || piece == 'R') {
        atomicAdd(&sharedRed, 1);
    } else if (piece == 'b' || piece == 'B') {
        atomicAdd(&sharedBlack, 1);
    }

    __syncthreads();

    if (local_tid == 0) {
        int result;
        if (sharedRed > sharedBlack) {
            result = -1;  // red wins
        } else if (sharedBlack > sharedRed) {
            result = 1;  // black wins
        } else {
            result = 0;  // tie
        }
        winner[boardIdx] = result;
    }
}

// Reduction kernel to aggregate results
__global__ void reduce(int *d_winners, int *d_finalResult, int BATCH_SIZE) {
    __shared__ int shared_data[THREADS_PER_BLOCK];
    
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Initialize the shared data array
    if (tid < BATCH_SIZE) {
        shared_data[threadIdx.x] = d_winners[tid];
    } else {
        shared_data[threadIdx.x] = 0; // out of bounds elements contribute 0
    }
    __syncthreads();

    // Parallel reduction within the block
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (threadIdx.x < stride && (tid + stride) < BATCH_SIZE) {
            shared_data[threadIdx.x] += shared_data[threadIdx.x + stride];
        }
        __syncthreads();
    }

    // Write the block's result into global memory
    if (threadIdx.x == 0) {
        atomicAdd(d_finalResult, shared_data[0]);
    }
}

// Function to run analysis
void runCudaAnalysis(BoardList *boards, int *likelihood) {
    const unsigned long long BATCH_SIZE = boards->count;

    BoardList *d_boards;
    hipMalloc(&d_boards, sizeof(BoardList));
    hipMemcpy(d_boards, boards, sizeof(BoardList), hipMemcpyHostToDevice);

    int *d_winners;
    hipMalloc(&d_winners, sizeof(int) * BATCH_SIZE);
    hipMemset(d_winners, 0, sizeof(int) * BATCH_SIZE);

    analyze<<<BATCH_SIZE, 64>>>(d_boards, d_winners);

    hipDeviceSynchronize();

    // Reduction ----------------------------------------------------------------------------------------------------------- 

    int *d_finalResult;
    hipMalloc(&d_finalResult, sizeof(int));
    hipMemset(d_finalResult, 0, sizeof(int));

    // Launch the reduction kernel
    int threadsPerBlock = 64;
    int numBlocks = (BATCH_SIZE + threadsPerBlock - 1) / threadsPerBlock;
    reduce<<<numBlocks, threadsPerBlock>>>(d_winners, d_finalResult, BATCH_SIZE);

    hipDeviceSynchronize();

    // Copy the final result back to host
    int finalResult = 0;
    hipMemcpy(&finalResult, d_finalResult, sizeof(int), hipMemcpyDeviceToHost);

    // Store the result in the output array (e.g., total wins)
    *likelihood = finalResult;

    // Free allocated memory
    hipFree(d_boards);
    hipFree(d_winners);
    hipFree(d_finalResult);
}
